#include "hip/hip_runtime.h"
/**
*	This is my first program in learning parallel programming using CUDA.
*	Equivalent to a hello World program :-)
*	This program basically performs two tasks:
*	1. It selects suitable CUDA enabled device(GPU) and prints the device properties
*	2. It demonstrate basic parallel addition of two arrays on the device(GPU) using add kernel.
*	Author: Shubham Singh
**/

#include "hip/hip_runtime.h"
#include <iostream>

#define N 10						/*N is size of arrays*/

using namespace std;

/************************************************************************************************************
*	Function:	Kernel to perform addition of two arrays in parallel on device(GPU)
*	Input:		Takes 3 pointer to int variables pointing to some memory locations on the device(GPU)
*	Output:		None
************************************************************************************************************/
__global__ void add(int *a, int *b, int *c)
{
	int i = blockIdx.x;				/*blockIDx.x holds ID of block and acts as index*/
	if (i < N)
		c[i] = a[i] + b[i];
}

int main()
{
	hipDeviceProp_t prop;			/*Structure variable to hold properties of a CUDA enabled device(GPU)*/
	int count, dev;	
	hipGetDevice(&dev);			/*Function to get current device ID and store device ID in dev*/
	cout << "ID of current cuda Device: " << dev<< endl;
	
	/*
	*	If system has multiple GPUs then
	*	Find CUDA device(GPU) having major computing capability greater than 1
	*	or having major computing capability 1 and minor computing capability greater than 3
	*/
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 3;
	hipChooseDevice(&dev, &prop);	/*Get device ID for revision greater than 1.3*/
	cout << "ID of CUDA device closest to revision 1.3: " << dev << endl;
	hipSetDevice(dev);				/*Set current device(GPU) to device ID dev which is having revision greater than 1.3*/
	
	/*
	*	Get properties of device dev from CUDA runtime and hold properties into prop
	*	and print few details of device dev
	*/
	hipGetDeviceProperties(&prop, dev);
	cout << "\n----Properties for device ID " << dev << "----" << endl << endl;
	cout << "Device name: " << prop.name << endl;
	cout << "Device clock rate(in kilohertz): " << prop.clockRate << endl;
	cout << "Device global memory(in bytes): " << prop.totalGlobalMem << endl;
	cout << "Device constant memory(in bytes): " << prop.totalConstMem << endl;
	if (prop.deviceOverlap)
		cout << "Device Overlap: Enabled" << endl;
	else
		cout << "Device Overlap: Disabled" << endl;
	if (prop.concurrentKernels)
		cout << "Concurrent kernels: Yes" << endl;
	else
		cout << "Concurrent kernels: Yes" << endl;
	cout << "Multiprocessor count: " << prop.multiProcessorCount << endl;
	cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
	cout << "Max thread dimension: " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << " " << endl;
	cout << "Max grid dimension: " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << " " << endl;
	cout << "Size of L2 cache(in bytes): " << prop.l2CacheSize << endl;
	cout << "Device's revision/compute capability: " << prop.major << "." << prop.minor << endl;
	
	/*
	*	So, we are done with printing device(GPU) details
	*	It's time to perform some parallel computation on device(GPU)
	*	having device ID dev.
	*	Let's perform simple array addition on device(GPU)
	*/
	cout << "\nLet's perform some addition on arrays" << endl;
	int *a, *b, *c;					/*Variables to hold arrays on host(CPU)*/
	int *dev_a, *dev_b, *dev_c;		/*Variables to hold arrays on device(GPU)*/

	/*Allocate memory for arrays on host(CPU)*/
	a = new int[N];
	b = new int[N];
	c = new int[N];

	/*Fill values in arrays on host(CPU)*/
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = N - i - 1;
	}

	/*Print arrays*/
	cout << "Array a: ";
	for (int i = 0; i < N; i++)
		cout << a[i] << " ";
	cout << "\nArray b: ";
	for (int i = 0; i < N; i++)
		cout << b[i] << " ";

	/*Allocate memory for arrays on device(GPU)*/
	hipMalloc((void **)&dev_a, N * sizeof(int));
	hipMalloc((void **)&dev_b, N * sizeof(int));
	hipMalloc((void **)&dev_c, N * sizeof(int));

	/*Copy arrays a and b from host(CPU) to device(GPU)*/
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	/*Call add kernel to perform addition on device(GPU)*/
	add <<< N, 1 >>> (dev_a, dev_b, dev_c);

	/*Copy c array back from device(GPU) to host(CPU)*/
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	/*Print sum*/
	cout << "\nArray c: ";
	for (int i = 0; i < N; i++)
		cout << c[i] << " ";
	cout << endl;

	/*Free memory allocated on host(CPU)*/
	delete[] a;
	delete[] b;
	delete[] c;

	/*Free memory allocated on device(GPU)*/
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}